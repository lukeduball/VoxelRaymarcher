#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>

#include <time.h>

#include "../geometry/VoxelCube.h"
#include "../geometry/VoxelSphere.h"

#include "../renderer/Renderer.cuh"
#include "../renderer/VoxelStructure.cuh"
#include "../renderer/images/ImageWriter.h"
#include "../renderer/camera/Camera.cuh"

#include "../cuckoohash/CuckooHashTable.cuh"

#define DEVICE_ID 0

__global__ void testLookupFunc(uint32_t* lookupValues, uint32_t numLookupKeys, CuckooHashTable* hashTable)
{
	for (uint32_t i = threadIdx.x; i < numLookupKeys; i += blockDim.x)
	{
		uint32_t result = hashTable->lookupValueForKey(lookupValues[i]);
		printf("Index:%d Key:%d -> Value:%d\n", i, lookupValues[i], result);
	}
}

int main()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	hipDeviceProp_t devProp;
	printf("Device Count: %d\n", devCount);
	if (devCount) 
	{
		hipSetDevice(DEVICE_ID);
		hipGetDeviceProperties(&devProp, DEVICE_ID);
	}
	printf("Device: %s\n", devProp.name);

	srand(time(NULL));

	uint32_t width = 1920;
	uint32_t height = 1080;
	float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

	Camera camera = Camera(Vector3(0.0f, 0.0f, 0.0f), Vector3(0.0f, 0.0f, -1.0f), Vector3(0.0f, 1.0f, 0.0f), 90.0f, aspectRatio);
	//copy the created camera to the GPU
	Camera* deviceCamera;
	hipMalloc(&deviceCamera, sizeof(Camera));
	hipMemcpy(deviceCamera, &camera, sizeof(Camera), hipMemcpyHostToDevice);

	ImageWriter imgWriter = ImageWriter();

	uint8_t* deviceFramebuffer;
	hipMalloc(&deviceFramebuffer, sizeof(uint8_t) * width * height * 3);

	std::unordered_map<uint32_t, uint32_t> voxelMap;
	//VoxelCube::generateVoxelCube(voxelMap, 512, 512, 512, 50);
	VoxelSphere::generateVoxelSphere(voxelMap, 32, 32, 32, 10);
	CuckooHashTable voxelHashTable = CuckooHashTable(voxelMap);

	//Hash table's GPU handle
	CuckooHashTable* deviceVoxelHashTable;
	//Move the hash table to the GPU
	hipMalloc(&deviceVoxelHashTable, sizeof(CuckooHashTable));
	hipMemcpy(deviceVoxelHashTable, &voxelHashTable, sizeof(CuckooHashTable), hipMemcpyHostToDevice);

	VoxelStructure voxelStructure = VoxelStructure(deviceVoxelHashTable, Vector3(-32.0f, -32.0f, -64.0f), 64);

	//Copy the voxel structure to the GPU
	VoxelStructure* deviceVoxelStructure;
	hipMalloc(&deviceVoxelStructure, sizeof(VoxelStructure));
	hipMemcpy(deviceVoxelStructure, &voxelStructure, sizeof(VoxelStructure), hipMemcpyHostToDevice);

	uint32_t numThreads = 8;
	dim3 blocks(width / numThreads + 1, height / numThreads + 1);
	dim3 threads(numThreads, numThreads);
	rayMarchScene <<<blocks, threads>>>(width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer);

	hipDeviceSynchronize();

	uint8_t* hostFramebuffer = static_cast<uint8_t*>(malloc(sizeof(uint8_t) * width * height * 3));
	hipMemcpy(hostFramebuffer, deviceFramebuffer, sizeof(uint8_t) * width * height * 3, hipMemcpyDeviceToHost);
	imgWriter.writeImage("output.png", hostFramebuffer, width, height, 3);

	free(hostFramebuffer);
	hipFree(deviceFramebuffer);
	hipFree(deviceCamera);

	hipFree(deviceVoxelStructure);
	hipFree(deviceVoxelHashTable);
		
	return EXIT_SUCCESS;
}