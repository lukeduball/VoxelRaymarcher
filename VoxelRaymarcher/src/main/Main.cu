#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>

#include "../geometry/VoxelCube.cuh"
#include "../geometry/VoxelFile.cuh"
#include "../geometry/VoxelFunctions.cuh"
#include "../geometry/VoxelSceneCPU.cuh"
#include "../geometry/VoxelSphere.cuh"

#include "../math/Random.cuh"
#include "../memory/MemoryUtils.h"

#include "../renderer/Renderer.cuh"
//#include "../renderer/OptimizedFunctions.cuh"
#include "../renderer/images/ImageWriter.h"
#include "../renderer/camera/Camera.cuh"

#include "../storage/CuckooHashTable.cuh"
#include "../storage/VoxelClusterStore.cuh"

#include <chrono>
#include <string>

#define DEVICE_ID 0

void setupConstantValues()
{
	Vector3f hostLightDirection = makeUnitVector(Vector3f(1.0f, 1.0f, 1.0f));
	hipMemcpyToSymbol(HIP_SYMBOL(LIGHT_DIRECTION), &hostLightDirection, sizeof(Vector3f));

	Vector3f hostLightColor = Vector3f(1.0f, 1.0f, 1.0f);
	hipMemcpyToSymbol(HIP_SYMBOL(LIGHT_COLOR), &hostLightColor, sizeof(Vector3f));

	Vector3f hostLightPosition = Vector3f(10.0f, 10.0f, -10.0f);
	hipMemcpyToSymbol(HIP_SYMBOL(LIGHT_POSITION), &hostLightPosition, sizeof(Vector3f));

	bool hostUsePointLight = false;
	hipMemcpyToSymbol(HIP_SYMBOL(USE_POINT_LIGHT), &hostUsePointLight, sizeof(bool));

	bool hostUseShadows = true;
	hipMemcpyToSymbol(HIP_SYMBOL(USE_SHADOWS), &hostUseShadows, sizeof(bool));
}

int32_t processStorageTypeCmdArg(int argc, char* argv[])
{
	if (argc > 2 && std::strcmp(argv[2], "hashtable") == 0)
	{
		std::cout << "Storage Type: Cuckoo Hash Table" << std::endl;
		return 1;
	}

	std::cout << "Storage Type: Voxel Cluster Storage" << std::endl;
	return 0;
}

int32_t processAlgorithmCmdArg(int argc, char* argv[])
{
	if (argc > 3 && std::strcmp(argv[3], "original") == 0)
	{
		std::cout << "Raymarching Algorithm: Original" << std::endl;
		return 1;
	}

	std::cout << "Raymarching Algorithm: Longest Axis" << std::endl;
	return 0;
}

int32_t processOptimizedCmdArg(int argc, char* argv[])
{
	if (argc > 3 && std::strcmp(argv[3], "optimized") == 0)
	{
		std::cout << "Running Optimized Functions" << std::endl;
		return true;
	}

	return false;
}

void pickCudaDevice()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	hipDeviceProp_t devProp;
	printf("Device Count: %d\n", devCount);
	if (devCount)
	{
		hipSetDevice(DEVICE_ID);
		hipGetDeviceProperties(&devProp, DEVICE_ID);
	}
	printf("Device: %s\n", devProp.name);
}

void populateVoxelScene(VoxelSceneCPU& voxelScene, StorageType storageType)
{
	//Read the voxel scene from the following file
	VoxelFile::readVoxelFile(voxelScene, "scene.vox");

	//Takes the CPU memory stored in standard containers and writes the data to the device
	voxelScene.generateVoxelScene(storageType);
}

void runRaymarchingKernel(uint32_t width, uint32_t height, bool useOptimizedFunctions, uint32_t rayMarchFunctionID, uint32_t voxelLookupFunctionID,
	Camera* deviceCameraPtr, VoxelSceneInfo* deviceSceneInfoPtr, uint8_t* deviceFramebufferPtr, StorageStructure** deviceScenePtr, uint32_t sceneArrayDiameter, int32_t minCoord)
{
	uint32_t numThreads = 8;
	dim3 blocks(width / numThreads + 1, height / numThreads + 1);
	dim3 threads(numThreads, numThreads);

	//Find the starting time for the clock
	auto startTime = std::chrono::high_resolution_clock::now();

	if (!useOptimizedFunctions)
	{
		if (rayMarchFunctionID == 0)
		{
			rayMarchSceneJumpAxis << <blocks, threads >> > (width, height, deviceCameraPtr, deviceSceneInfoPtr, deviceFramebufferPtr,
				deviceScenePtr, sceneArrayDiameter, minCoord);
		}
		else if (rayMarchFunctionID == 1)
		{
			rayMarchSceneOriginal << <blocks, threads >> > (width, height, deviceCameraPtr, deviceSceneInfoPtr, deviceFramebufferPtr,
				deviceScenePtr, sceneArrayDiameter, minCoord);
		}
	}
	else
	{
		std::cout << "Optimized functions are currently disabled" << std::endl;
		//Jump Axis with VCS
		if (rayMarchFunctionID == 0 && voxelLookupFunctionID == 0)
		{
			//rayMarchSceneJumpAxisVCS << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceVoxelClusterStore);
		}
		//Original with VCS
		else if (rayMarchFunctionID == 1 && voxelLookupFunctionID == 0)
		{
			//rayMarchSceneOriginalVCS << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceVoxelClusterStore);
		}
		//Jump Axis with Cuckoo Hash Table
		else if (rayMarchFunctionID == 0 && voxelLookupFunctionID == 1)
		{
			//rayMarchSceneJumpAxisHashTable << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceHashTable);
		}
		//Original with Cuckoo Hash Table
		else
		{
			//rayMarchSceneOriginalHashTable << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceHashTable);
		}
	}
	hipError_t err = hipPeekAtLastError();
	std::cout << hipGetErrorString(err) << std::endl;

	hipDeviceSynchronize();

	auto endTime = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
	std::cout << "Execution Time for Ray Marching Algorithm is: " << duration.count() << " microseconds" << std::endl;
}

void writeResultingImageToDisk(uint32_t width, uint32_t height, uint8_t* deviceFramebufferPtr)
{
	uint8_t* hostFramebuffer = static_cast<uint8_t*>(malloc(sizeof(uint8_t) * width * height * 3));
	hipMemcpy(hostFramebuffer, deviceFramebufferPtr, sizeof(uint8_t) * width * height * 3, hipMemcpyDeviceToHost);
	ImageWriter imgWriter = ImageWriter();
	imgWriter.writeImage("output.png", hostFramebuffer, width, height, 3);

	//Clean up the host framebuffer memory after writing to the disk
	free(hostFramebuffer);
}

int main(int argc, char* argv[])
{
	Random::initialize();

	//Setup the command line arguments
	if (argc <= 1)
		std::cout << "You need to provide a voxel scale" << std::endl;
	int32_t scale = std::stoi(argv[1]);
	int32_t voxelLookupFunctionID = processStorageTypeCmdArg(argc, argv);
	int32_t rayMarchFunctionID = processAlgorithmCmdArg(argc, argv);
	bool useOptimizedFunctions = processOptimizedCmdArg(argc, argv);

	pickCudaDevice();
	//Pass values that won't change during a kernel call to the GPU
	setupConstantValues();

	uint32_t width = 1920;
	uint32_t height = 1080;
	float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

	Camera camera = Camera(Vector3f(6.0f, 2.0f, 6.0f), Vector3f(0.0f, 0.0f, -1.0f), Vector3f(0.0f, 1.0f, 0.0f), 60.0f, aspectRatio);
	//copy the created camera to the GPU
	CudaDeviceMemoryJanitor<Camera> deviceCameraJanitor(&camera, "Camera Memory");

	//Allocate the memory on the device that will hold the resulting image
	CudaDeviceMemoryJanitor<uint8_t> deviceFramebufferJanitor(width * height * 3, "Framebuffer Memory");

	VoxelSceneCPU voxelScene;
	//Populate the passed Voxel Scene with voxel data
	populateVoxelScene(voxelScene, StorageType(voxelLookupFunctionID));
	CudaDeviceMemoryJanitor<StorageStructure*> deviceVoxelSceneJanitor(voxelScene.getArraySize(), "Voxel Scene Memory (table of pointers into region's storage structure)");
	//Kernel call to populate the voxel scene on the device. This call needs to happen because it creates virtual classes which need to be made on the device
	generateVoxelScene<<<1, 1>>>(deviceVoxelSceneJanitor.devicePtr, voxelScene.deviceVoxelScene, voxelScene.getArraySize(), StorageType(voxelLookupFunctionID));
	//Wait for the previous kernel call to finish
	hipDeviceSynchronize();
	
	VoxelSceneInfo voxelSceneInfo = VoxelSceneInfo(Vector3f(0.0f, 0.0f, 0.0f), scale);
	CudaDeviceMemoryJanitor<VoxelSceneInfo> deviceVoxelSceneInfoJanitor(&voxelSceneInfo, "Voxel Scene Info Memory");

	//Run the raymarching kernel with the specified options and scene
	runRaymarchingKernel(width, height, useOptimizedFunctions, rayMarchFunctionID, voxelLookupFunctionID, deviceCameraJanitor.devicePtr,
		deviceVoxelSceneInfoJanitor.devicePtr, deviceFramebufferJanitor.devicePtr, deviceVoxelSceneJanitor.devicePtr, voxelScene.getArrayDiameter(), voxelScene.getMinCoord());

	//Get the resulting image from the device and output it to the disk
	writeResultingImageToDisk(width, height, deviceFramebufferJanitor.devicePtr);

	//Clean up the memory on the device from the voxel scene
	voxelScene.cleanupVoxelScene();
		
	return EXIT_SUCCESS;
}