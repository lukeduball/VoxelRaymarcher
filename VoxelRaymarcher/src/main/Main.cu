﻿#include <hip/hip_runtime.h>

#include "../geometry/VoxelCube.cuh"
#include "../geometry/VoxelFunctions.cuh"
#include "../geometry/VoxelSphere.cuh"

#include "../renderer/Renderer.cuh"
#include "../renderer/OptimizedFunctions.cuh"
#include "../renderer/VoxelStructure.cuh"
#include "../renderer/images/ImageWriter.h"
#include "../renderer/camera/Camera.cuh"

#include "../storage/CuckooHashTable.cuh"
#include "../storage/VoxelClusterStore.cuh"

#include <string>

#define DEVICE_ID 0

int main(int argc, char* argv[])
{
	int32_t voxelLookupFunctionID = 0;
	int32_t rayMarchFunctionID = 0;
	bool useOptimizedFunctions = false;

	//If there are command line args, setup the commands
	if (argc == 3)
	{
		if (std::strcmp(argv[1], "hashtable") == 0)
		{
			voxelLookupFunctionID = 1;
			std::cout << "Storage Type: Cuckoo Hash Table" << std::endl;
		}
		else
		{
			std::cout << "Storage Type: Voxel Cluster Storage" << std::endl;
		}
		if (std::strcmp(argv[2], "original") == 0)
		{
			rayMarchFunctionID = 1;
			std::cout << "Raymarch Algorithm: Original" << std::endl;
		}
		else
		{
			std::cout << "Raymarch Algorithm: Jump Axis" << std::endl;
		}
	}
	else if (argc == 4)
	{
		useOptimizedFunctions = true;
		if (std::strcmp(argv[2], "hashtable") == 0)
		{
			voxelLookupFunctionID = 1;
			std::cout << "Storage Type: Cuckoo Hash Table" << std::endl;
		}
		else
		{
			std::cout << "Storage Type: Voxel Cluster Store" << std::endl;
		}
		if (std::strcmp(argv[3], "original") == 0)
		{
			rayMarchFunctionID = 1;
			std::cout << "Raymarch Algorithm: Original" << std::endl;
		}
		else
		{
			std::cout << "Raymarch Algorithm: Jump Axis" << std::endl;
		}
	}
	else
	{
		std::cout << "Using Default Arguments!" << std::endl;
	}

	int devCount;
	hipGetDeviceCount(&devCount);
	hipDeviceProp_t devProp;
	printf("Device Count: %d\n", devCount);
	if (devCount) 
	{
		hipSetDevice(DEVICE_ID);
		hipGetDeviceProperties(&devProp, DEVICE_ID);
	}
	printf("Device: %s\n", devProp.name);

	uint32_t width = 1920;
	uint32_t height = 1080;
	float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

	Camera camera = Camera(Vector3(0.0f, 0.0f, 0.0f), Vector3(0.0f, 0.0f, -1.0f), Vector3(0.0f, 1.0f, 0.0f), 90.0f, aspectRatio);
	//copy the created camera to the GPU
	Camera* deviceCamera;
	hipMalloc(&deviceCamera, sizeof(Camera));
	hipMemcpy(deviceCamera, &camera, sizeof(Camera), hipMemcpyHostToDevice);

	uint8_t* deviceFramebuffer;
	hipMalloc(&deviceFramebuffer, sizeof(uint8_t) * width * height * 3);

	std::unordered_map<uint32_t, uint32_t> voxelMap;
	//VoxelCube::generateVoxelCube(voxelMap, 512, 512, 512, 50);
	VoxelSphere::generateVoxelSphere(voxelMap, BLOCK_SIZE / 2, BLOCK_SIZE / 2, BLOCK_SIZE / 2, BLOCK_SIZE / 6);

	//Create the GPU handles for both storage types
	CuckooHashTable* deviceHashTable = nullptr;
	VoxelClusterStore* deviceVoxelClusterStore = nullptr;

	//Case for the Voxel Cluster Store being the storage type
	if (voxelLookupFunctionID == 0)
	{
		//Generate the voxel cluster store on the CPU
		VoxelClusterStore voxelClusterStore = VoxelClusterStore(voxelMap);

		//Move the voxel cluster store to the GPU
		hipMalloc(&deviceVoxelClusterStore, sizeof(VoxelClusterStore));
		hipMemcpy(deviceVoxelClusterStore, &voxelClusterStore, sizeof(VoxelClusterStore), hipMemcpyHostToDevice);
	}
	//Case for the Cuckoo Hash table being the storage type
	else
	{
		//Generate the cuckoo hash table on the CPU
		CuckooHashTable voxelHashTable = CuckooHashTable(voxelMap);

		//Move the hash table to the GPU
		hipMalloc(&deviceHashTable, sizeof(CuckooHashTable));
		hipMemcpy(deviceHashTable, &voxelHashTable, sizeof(CuckooHashTable), hipMemcpyHostToDevice);
	}
	
	VoxelStructure voxelStructure = VoxelStructure(Vector3(-((float)BLOCK_SIZE / 2), -((float)BLOCK_SIZE / 2), -(float)BLOCK_SIZE), BLOCK_SIZE);

	//Copy the voxel structure to the GPU
	VoxelStructure* deviceVoxelStructure;
	hipMalloc(&deviceVoxelStructure, sizeof(VoxelStructure));
	hipMemcpy(deviceVoxelStructure, &voxelStructure, sizeof(VoxelStructure), hipMemcpyHostToDevice);

	uint32_t numThreads = 8;
	dim3 blocks(width / numThreads + 1, height / numThreads + 1);
	dim3 threads(numThreads, numThreads);

	if (!useOptimizedFunctions)
	{
		if (rayMarchFunctionID == 0)
		{
			rayMarchSceneJumpAxis << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceVoxelClusterStore, deviceHashTable);
		}
		else if (rayMarchFunctionID == 1)
		{
			rayMarchSceneOriginal << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceVoxelClusterStore, deviceHashTable);
		}
	}
	else
	{
		//Jump Axis with VCS
		if (rayMarchFunctionID == 0 && voxelLookupFunctionID == 0)
		{
			//rayMarchSceneJumpAxisVCS << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceVoxelClusterStore);
		}
		//Original with VCS
		else if (rayMarchFunctionID == 1 && voxelLookupFunctionID == 0)
		{
			rayMarchSceneOriginalVCS << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceVoxelClusterStore);
		}
		//Jump Axis with Cuckoo Hash Table
		else if (rayMarchFunctionID == 0 && voxelLookupFunctionID == 1)
		{
			rayMarchSceneJumpAxisHashTable << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceHashTable);
		}
		//Original with Cuckoo Hash Table
		else
		{
			rayMarchSceneOriginalHashTable << <blocks, threads >> > (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, deviceHashTable);
		}
	}
	hipError_t err = hipPeekAtLastError();
	std::cout << hipGetErrorString(err) << std::endl;

	hipDeviceSynchronize();


	uint8_t* hostFramebuffer = static_cast<uint8_t*>(malloc(sizeof(uint8_t) * width * height * 3));
	hipMemcpy(hostFramebuffer, deviceFramebuffer, sizeof(uint8_t) * width * height * 3, hipMemcpyDeviceToHost);
	ImageWriter imgWriter = ImageWriter();
	imgWriter.writeImage("output.png", hostFramebuffer, width, height, 3);

	free(hostFramebuffer);
	hipFree(deviceFramebuffer);
	hipFree(deviceCamera);

	hipFree(deviceVoxelStructure);

	if(deviceHashTable)
		hipFree(deviceHashTable);
	if(deviceVoxelClusterStore)
		hipFree(deviceVoxelClusterStore);
		
	return EXIT_SUCCESS;
}