#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>

#include "../geometry/VoxelCube.cuh"
#include "../geometry/VoxelSphere.cuh"

#include "../renderer/Renderer.cuh"
#include "../renderer/VoxelStructure.cuh"
#include "../renderer/images/ImageWriter.h"
#include "../renderer/camera/Camera.cuh"

#include "../cuckoohash/CuckooHashTable.cuh"
#include "../voxelclusterstore/VoxelClusterStore.cuh"

#include <string>

#define DEVICE_ID 0

__global__ void testLookupFunc(uint32_t* lookupValues, uint32_t numLookupKeys, CuckooHashTable* hashTable)
{
	for (uint32_t i = threadIdx.x; i < numLookupKeys; i += blockDim.x)
	{
		uint32_t result = hashTable->lookupValueForKey(lookupValues[i]);
		printf("Index:%d Key:%d -> Value:%d\n", i, lookupValues[i], result);
	}
}

int main(int argc, char* argv[])
{
	int32_t voxelLookupFunctionID = 0;
	int32_t rayMarchFunctionID = 0;

	//If there are command line args, setup the commands
	if (argc == 3)
	{
		if (std::strcmp(argv[1], "hashtable") == 0)
		{
			voxelLookupFunctionID = 1;
			std::cout << "Storage Type: Cuckoo Hash Table" << std::endl;
		}
		else
		{
			std::cout << "Storage Type: Voxel Cluster Storage" << std::endl;
		}
		if (std::strcmp(argv[2], "original") == 0)
		{
			rayMarchFunctionID = 1;
			std::cout << "Raymarch Algorithm: Original" << std::endl;
		}
		else
		{
			std::cout << "Raymarch Algorithm: Jump Axis" << std::endl;
		}
	}
	else
	{
		std::cout << "Using Default Arguments!" << std::endl;
	}

	int devCount;
	hipGetDeviceCount(&devCount);
	hipDeviceProp_t devProp;
	printf("Device Count: %d\n", devCount);
	if (devCount) 
	{
		hipSetDevice(DEVICE_ID);
		hipGetDeviceProperties(&devProp, DEVICE_ID);
	}
	printf("Device: %s\n", devProp.name);

	uint32_t width = 1920;
	uint32_t height = 1080;
	float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

	Camera camera = Camera(Vector3(0.0f, 0.0f, 0.0f), Vector3(0.0f, 0.0f, -1.0f), Vector3(0.0f, 1.0f, 0.0f), 90.0f, aspectRatio);
	//copy the created camera to the GPU
	Camera* deviceCamera;
	hipMalloc(&deviceCamera, sizeof(Camera));
	hipMemcpy(deviceCamera, &camera, sizeof(Camera), hipMemcpyHostToDevice);

	uint8_t* deviceFramebuffer;
	hipMalloc(&deviceFramebuffer, sizeof(uint8_t) * width * height * 3);

	std::unordered_map<uint32_t, uint32_t> voxelMap;
	//VoxelCube::generateVoxelCube(voxelMap, 512, 512, 512, 50);
	VoxelSphere::generateVoxelSphere(voxelMap, 32, 32, 32, 10);

	//Create the GPU handles for both storage types
	CuckooHashTable* deviceVoxelHashTable = nullptr;
	VoxelClusterStore* deviceVoxelClusterStore = nullptr;

	VoxelStructure voxelStructure;

	//Case for the Voxel Cluster Store being the storage type
	if (voxelLookupFunctionID == 0)
	{
		//Generate the voxel cluster store on the CPU
		VoxelClusterStore voxelClusterStore = VoxelClusterStore(voxelMap);

		//Move the voxel cluster store to the GPU
		hipMalloc(&deviceVoxelClusterStore, sizeof(VoxelClusterStore));
		hipMemcpy(deviceVoxelClusterStore, &voxelClusterStore, sizeof(VoxelClusterStore), hipMemcpyHostToDevice);

		voxelStructure = VoxelStructure(deviceVoxelClusterStore, Vector3(-32.0f, -32.0f, -64.0f), 64);
	}
	//Case for the Cuckoo Hash table being the storage type
	else
	{
		//Generate the cuckoo hash table on the CPU
		CuckooHashTable voxelHashTable = CuckooHashTable(voxelMap);

		//Move the hash table to the GPU
		hipMalloc(&deviceVoxelHashTable, sizeof(CuckooHashTable));
		hipMemcpy(deviceVoxelHashTable, &voxelHashTable, sizeof(CuckooHashTable), hipMemcpyHostToDevice);

		voxelStructure = VoxelStructure(deviceVoxelHashTable, Vector3(-32.0f, -32.0f, -64.0f), 64);
	}
	

	//Copy the voxel structure to the GPU
	VoxelStructure* deviceVoxelStructure;
	hipMalloc(&deviceVoxelStructure, sizeof(VoxelStructure));
	hipMemcpy(deviceVoxelStructure, &voxelStructure, sizeof(VoxelStructure), hipMemcpyHostToDevice);

	uint32_t numThreads = 8;
	dim3 blocks(width / numThreads + 1, height / numThreads + 1);
	dim3 threads(numThreads, numThreads);
	rayMarchScene <<<blocks, threads>>> (width, height, deviceCamera, deviceVoxelStructure, deviceFramebuffer, rayMarchFunctionID, voxelLookupFunctionID);

	hipDeviceSynchronize();


	uint8_t* hostFramebuffer = static_cast<uint8_t*>(malloc(sizeof(uint8_t) * width * height * 3));
	hipMemcpy(hostFramebuffer, deviceFramebuffer, sizeof(uint8_t) * width * height * 3, hipMemcpyDeviceToHost);
	ImageWriter imgWriter = ImageWriter();
	imgWriter.writeImage("output.png", hostFramebuffer, width, height, 3);

	free(hostFramebuffer);
	hipFree(deviceFramebuffer);
	hipFree(deviceCamera);

	hipFree(deviceVoxelStructure);

	if(deviceVoxelHashTable)
		hipFree(deviceVoxelHashTable);
	if (deviceVoxelClusterStore)
		hipFree(deviceVoxelClusterStore);
		
	return EXIT_SUCCESS;
}